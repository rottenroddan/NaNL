#include "hip/hip_runtime.h"
//
// Created by steve on 11/27/2022.
//
#include "Matrix.cuh"

template<typename T>
NaNL::Matrix<T, NaNL::Device::Host>::Matrix(unsigned long numberOfRows, unsigned long numberOfCols) :
        BaseMatrix<T>(numberOfRows, numberOfCols, _freePagedMemory) {
#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_START;
#endif

    _allocateMemory(numberOfRows, numberOfCols);

#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_END;
#endif
}

template<typename T>
NaNL::Matrix<T, NaNL::Device::Host>::Matrix(const NaNL::Matrix<T, NaNL::Device::Host> &copyMatrix) noexcept :
BaseMatrix<T>(copyMatrix.rows, copyMatrix.cols, _freePagedMemory) { ; }



template<typename T>
NaNL::Matrix<T, NaNL::Device::Host>::Matrix(NaNL::Matrix<T, NaNL::Device::Host> &&copyMatrix)  noexcept :
BaseMatrix<T>(copyMatrix.rows, copyMatrix.cols, _freePagedMemory) {
#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_START;
#endif

    this->matrix = std::move(copyMatrix.matrix);

#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_END;
#endif
}

template<typename T>
void NaNL::Matrix<T, NaNL::Device::Host>::_allocateMemory(unsigned long rows, unsigned long cols)
{
#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_START;
#endif

    T* _pagedArr = new T[this->totalSize];
    this->matrix = std::unique_ptr<T[], void(*)(T*)>(_pagedArr, _freePagedMemory);

#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_END;
#endif

}

template<typename T>
void NaNL::Matrix<T, NaNL::Device::Host>::_hostAddMatrices(T* _a, T* _b, T* _c, unsigned long _blockSize, unsigned long _offset) {
    for(unsigned long i = 0; i < _blockSize; i++) {
        _c[_offset + i] = _a[_offset + i] + _b[_offset + i];
    }
}

template<typename T>
void NaNL::Matrix<T, NaNL::Device::Host>::add(const NaNL::BaseMatrix<T> &bMatrix) {
#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_START;
#endif

    if(this->validateMatricesAreSameShape(bMatrix)) {
        //throw new MatrixIsInvalidShape();
    }

    T* a = this->matrix.get();
    T* b = bMatrix.matrix.get();
    T* c = this->matrix.get();

    // get total totalThreads.
    unsigned long totalThreads = NaNL::ThreadPool::getInstance()->getAllocatedThreads();

    // calculate block size per thread.
    unsigned long blockSize = this->totalSize / totalThreads;
    unsigned long remainder = this->totalSize - blockSize * totalThreads;
    unsigned long threadOffset = 0;

    NaNL::ThreadPool* threadPool = NaNL::ThreadPool::getInstance();
    std::deque<std::future<void>> results;

    for(unsigned long i = 0; i < totalThreads; i++) {
        unsigned long modifiedBlockSize = (remainder == 0) ? blockSize : blockSize + 1;
        std::future<void> result = threadPool->queue([this, a, b, c, modifiedBlockSize, threadOffset] { _hostAddMatrices(a,b,c, modifiedBlockSize, threadOffset); });
        results.push_back(std::move(result));
        threadOffset += modifiedBlockSize;

        if(remainder != 0) {
            remainder--;
        }
    }

    // wait on all futures to be populated/deferred.
    for(auto & result : results) {
        result.wait();
    }

#ifdef PERFORMANCE_LOGGING
    PERFORMANCE_LOGGING_END;
#endif
}

template<typename T>
NaNL::Matrix<T, NaNL::Device::Host> &
NaNL::Matrix<T, NaNL::Device::Host>::operator=(const NaNL::Matrix<T, NaNL::Device::Host> &rhs) {
    return *this;
}





