//
// Created by steve on 5/29/2023.
//

#include "PinnedMemoryBlock.cuh"

namespace NaNL {
    template<class T>
    NaNL::PinnedMemoryBlock<T>::PinnedMemoryBlock(uint64_t totalSize) : _matrix(nullptr, nullptr)
    {
        T *_pinnedArr;
        gpuErrchk(hipHostMalloc((void **) &_pinnedArr, totalSize * sizeof(T)));
        this->_matrix = std::unique_ptr<T[], void (*)(T*)>(_pinnedArr, _freePinnedMemory);
    }
} // NaNL