#include "hip/hip_runtime.h"
//
// Created by steve on 2/26/2023.
//

#include <iostream>
#include <PinnedMemoryBlock.cuh>
#include <PagedMemoryBlock.cuh>
#include <DeviceMemoryBlock.cuh>
#include <Matrix.cuh>
#include <MatrixFileLoader.cuh>
#include <MatrixOutBinaryFileLoader.cuh>
#include <MatrixInBinaryFileLoader.cuh>
#include <TensorCoreAligned32.cuh>
#include <type_traits>

#include <math.h>

float p2p_copy (size_t size)
{
    int *pointers[2];

    hipSetDevice (0);
    hipDeviceEnablePeerAccess (1, 0);
    hipMalloc (&pointers[0], size);

    hipSetDevice (1);
    hipDeviceEnablePeerAccess (0, 0);
    hipMalloc (&pointers[1], size);

    hipEvent_t begin, end;
    hipEventCreate (&begin);
    hipEventCreate (&end);

    hipEventRecord (begin);
    hipMemcpyAsync (pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
    hipEventRecord (end);
    hipEventSynchronize (end);

    float elapsed;
    hipEventElapsedTime (&elapsed, begin, end);
    elapsed /= 1000;

    hipSetDevice (0);
    hipFree (pointers[0]);

    hipSetDevice (1);
    hipFree (pointers[1]);

    hipEventDestroy (end);
    hipEventDestroy (begin);

    return elapsed;
}

void test() {
    std::vector<std::thread> threads;

    hipSetDevice(0);
    NaNL::Matrix<int, NaNL::DeviceMemoryBlock, NaNL::Unaligned> A(10000,10000);
    auto x = A.copyTo<NaNL::DeviceMemoryBlock, NaNL::Unaligned>();



    p2p_copy(4000);

    threads.emplace_back([&] {
        // cuda initializer
        hipSetDevice(0);
        hipDeviceEnablePeerAccess(1, 0);
        NaNL::Matrix<int, NaNL::PinnedMemoryBlock, NaNL::Unaligned> m(10000, 10000);

        DWORD threadId = GetCurrentThreadId();
        NaNL::Logger* logger = NaNL::Logger::GetInstance();
        logger->begin(threadId, "3080-Ti", "");

        for(uint64_t i = 0; i < 100; i++) {
            m.add<NaNL::PinnedMemoryBlock, NaNL::Unaligned>(A, NaNL::MatrixAddOperation::Cuda);
        }

        logger->end(threadId);
        logger->log(threadId);
    });

    threads.emplace_back([&] {
        // cuda initializer
        hipSetDevice(1);
        hipDeviceEnablePeerAccess(0, 0);
        NaNL::Matrix<int, NaNL::PinnedMemoryBlock, NaNL::Unaligned> m(10000, 10000);

        DWORD threadId = GetCurrentThreadId();
        NaNL::Logger* logger = NaNL::Logger::GetInstance();
        logger->begin(threadId, "2080-Ti", "");

        for(uint64_t i = 0; i < 100; i++) {
            m.add<NaNL::PinnedMemoryBlock, NaNL::Unaligned>(A, NaNL::MatrixAddOperation::Cuda);
        }

        logger->end(threadId);
        logger->log(threadId);
    });


    for(auto& thread : threads) {
        thread.join();
    }
}

int main() {
    //test();




    double k = 200.0;
    std::cout << sin(k) << std::endl;

    DWORD threadId = GetCurrentThreadId();
    NaNL::Logger* logger = NaNL::Logger::GetInstance();
    logger->begin(threadId, "File Loader test", "");

    NaNL::Matrix<uint64_t, NaNL::PagedMemoryBlock, NaNL::TensorCoreAligned32> u(2,2);
    u[0][0] = 5;
    u[0][1] = 9;
    u[1][0] = 4;
    u[1][1] = 3;

    auto c = u.add<NaNL::PagedMemoryBlock, NaNL::Unaligned>(u);

    for(uint64_t i = 0; i < c.getRows(); i++) {
        for(uint64_t j = 0; j < c.getCols(); j++) {
            std::cout << c[i][j] << std::endl;
        }
    }




   // test(aa);
    //test(bb);

    //static_assert(is_base<Base<int>, A<int>>::value, "If error: A is not derived from Base");

//    A<int> aa;
//    test(aa);

    //B bb;
    //test(bb);

//    A a(10);
//    A b(5);
//
//    b = std::move(a);
//    std::cout << b.x << std::endl;

   // std::cout << sizeof(size_t) << std::endl;

//    NaNL::Matrix<int, NaNL::PagedMemoryBlock, NaNL::TensorCoreAligned32> x(100, 100);
//
//    NaNL::Matrix<int, NaNL::PagedMemoryBlock, NaNL::Unaligned> y(100, 100);
//
//    std::cout << y[0][100] << std::endl;


//    NaNL::Matrix<int, NaNL::PinnedMemoryBlock, NaNL::Unaligned> b(100, 100);
//    NaNL::Matrix<int, NaNL::DeviceMemoryBlock, NaNL::Unaligned> d(100, 100);
//
//    //NaNL::Matrix<int, NaNL::PinnedMemoryBlock, NaNL::Unaligned>::addHost<>(b, d);
//
//    NaNL::Matrix<int, NaNL::PagedMemoryBlock, NaNL::Unaligned> a(100,100);
//    NaNL::Matrix<float, NaNL::PagedMemoryBlock, NaNL::Unaligned> aa(100,100);
//    NaNL::Matrix<float, NaNL::PagedMemoryBlock, NaNL::Unaligned> aaa(100,100);
//
//    NaNL::Matrix<double, NaNL::PagedMemoryBlock, NaNL::Unaligned> jjj(200,200);
//
//
//    b.getMatrix();
//
//    std::cout << a[0][0] << std::endl;

   // matrixTest(a);
   // matrixTest(b);
   // matrixTest(d);
    //matrixTest(d);

    //a.add<NaNL::PagedMemoryBlock, NaNL::Unaligned>(b);


//    std::chrono::time_point<std::chrono::high_resolution_clock> pagedStart, pagedEnd, pinnedStart, pinnedEnd;
//    std::chrono::time_point<std::chrono::high_resolution_clock> pagedAllocStart, pagedAllocEnd, pinnedAllocStart, pinnedAllocEnd, pinnedAlloc2Start, pinnedAlloc2End;
//
//    uint64_t cols = 4000000;
//
//    pagedAllocStart = std::chrono::high_resolution_clock::now();
//    int64_t* _pagedArr = new int64_t[cols];
//    pagedAllocEnd = std::chrono::high_resolution_clock::now();
//
//    int64_t* _pinnedArr = nullptr;
//    int64_t* _pinnedArrTemp = nullptr;
//
//    pinnedAllocStart = std::chrono::high_resolution_clock::now();
//    hipHostMalloc((void**)&_pinnedArr, cols*sizeof(uint64_t));
//    pinnedAllocEnd = std::chrono::high_resolution_clock::now();
//
//    pinnedAlloc2Start = std::chrono::high_resolution_clock::now();
//    hipHostMalloc((void**)&_pinnedArrTemp, cols*sizeof(uint64_t));
//    pinnedAlloc2End = std::chrono::high_resolution_clock::now();
//
//    pagedStart = std::chrono::high_resolution_clock::now();
//    for(uint64_t i = 0; i < cols; i++) {
//        _pagedArr[i] = i*i;
//    }
//    pagedEnd = std::chrono::high_resolution_clock::now();
//
//
//
//    pinnedStart = std::chrono::high_resolution_clock::now();
//    for(uint64_t i = 0; i < cols; i++) {
//        _pinnedArr[i] = i*i;
//    }
//    pinnedEnd = std::chrono::high_resolution_clock::now();
//
//    std::cout << "Paged Alloc time:      " << (pagedAllocEnd - pagedAllocStart).count() << "ms." << std::endl;
//    std::cout << "Pinned Alloc time:     " << (pinnedAllocEnd - pinnedAllocStart).count() << "ms." << std::endl;
//    std::cout << "2nd Pinned Alloc time: " << (pinnedAlloc2End - pinnedAlloc2Start).count() << "ms." << std::endl;
//    std::cout << "Paged time:  " << (pagedEnd - pagedStart).count() << "ms." << std::endl;
//    std::cout << "Pinned time: " << (pinnedEnd - pinnedStart).count() << "ms." << std::endl;

//    uint64_t rows = 10;
//    uint64_t cols = 10;
//
//    NaNL::Matrix<int> p(rows, cols);
//
//    for(int i = 0; i < rows; i++) {
//        for(int j = 0; j < 10; j++) {
//            p[i][j] = i * j;
//        }
//    }
//
//    int *hostPtr;
//    int *devicePtr;
//
//    gpuErrchk(hipSetDevice(0));
//    gpuErrchk(hipMalloc(&devicePtr, sizeof(int)));
//
//    hipPointerAttribute_t attribute;
//    hipPointerGetAttributes(&attribute, devicePtr);
//
//    if(attribute.type == hipMemoryType::hipMemoryTypeHost) {
//
//    }
//
//    auto copyM = p.copyTo<NaNL::PinnedMemoryBlock, NaNL::Unaligned>();
//
//    auto x = p.add<NaNL::PagedMemoryBlock, NaNL::Unaligned>(p, NaNL::MatrixDeviceOperation::Host);
//
//
//    std::cout << copyM.getTotalSize() << std::endl;
//    std::cout << x.getTotalSize() << std::endl;
//
//    for(int i = 0; i < rows; i++) {
//        for(int j = 0; j < cols; j++) {
//            std::cout << copyM[i][j] << std::endl;
//        }
//    }

    //NaNL::Unaligned<int, NaNL::PagedMemoryBlock> un(100, 100);

    //BaseMatrix<int> a(100 , 100);

//    hipFree(_pinnedArr);
//    hipFree(_pinnedArrTemp);

    return 0;
}