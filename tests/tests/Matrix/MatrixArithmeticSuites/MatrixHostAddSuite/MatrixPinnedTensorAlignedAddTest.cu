//
// Created by steve on 6/6/2023.
//

#include "gtest/gtest.h"
#include <hip/hip_fp16.h>
#include <Matrix.cuh>
#include "MatrixHostAddSuite.cuh"
#include <TensorCoreAligned16.cuh>

TEST_F(MatrixHostAddSuite, Should_Add_Small_Matrices_To_Correct_Values_When_Pinned_Tensor_Aligned_16) {
    try {
        auto a = smallTestMatrices->getCopyOfA<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();
        auto b = smallTestMatrices->getCopyOfB<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();
        auto truth = smallTestMatrices->getCopyOfTruth<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();

        auto c = a.add<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>(b, NaNL::MatrixDeviceOperation::Host);

        ASSERT_EQ(c.getRows(), truth.getRows());
        ASSERT_EQ(c.getCols(), truth.getCols());

        for (unsigned int i = 0; i < c.getRows(); i++) {
            for (unsigned int j = 0; j < c.getCols(); j++) {
                ASSERT_EQ(c[i][j], truth[i][j]);
            }
        }
    } catch (std::exception &e) {
        std::cout << e.what();
        FAIL();
    }
}

TEST_F(MatrixHostAddSuite, Should_Add_Medium_Matrices_To_Correct_Values_When_Pinned_Tensor_Aligned_16) {
    try {
        auto a = mediumTestMatrices->getCopyOfA<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();;
        auto b = mediumTestMatrices->getCopyOfB<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();;
        auto truth = mediumTestMatrices->getCopyOfTruth<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();;

        auto c = a.add<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>(b, NaNL::MatrixDeviceOperation::Host);

        ASSERT_EQ(c.getRows(), truth.getRows());
        ASSERT_EQ(c.getCols(), truth.getCols());

        for (unsigned int i = 0; i < c.getRows(); i++) {
            for (unsigned int j = 0; j < c.getCols(); j++) {
                ASSERT_EQ(c[i][j], truth[i][j]);
            }
        }
    } catch (std::exception &e) {
        std::cout << e.what();
        FAIL();
    }
}

TEST_F(MatrixHostAddSuite, Should_Add_Large_Matrices_To_Correct_Values_When_Pinned_Tensor_Aligned_16) {
    try {
        auto a = largeTestMatrices->getCopyOfA<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();;
        auto b = largeTestMatrices->getCopyOfB<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();;
        auto truth = largeTestMatrices->getCopyOfTruth<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>().copyTo<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned16, int16_t>();;

        auto c = a.add<NaNL::PinnedMemoryBlock, NaNL::TensorCoreAligned8>(b);

        ASSERT_EQ(c.getRows(), truth.getRows());
        ASSERT_EQ(c.getCols(), truth.getCols());

        for (unsigned int i = 0; i < c.getRows(); i++) {
            for (unsigned int j = 0; j < c.getCols(); j++) {
                ASSERT_EQ(c[i][j], truth[i][j]);
            }
        }
    } catch (std::exception &e) {
        std::cout << e.what();
        FAIL();
    }
}